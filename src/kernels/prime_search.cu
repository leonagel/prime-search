#include "hip/hip_runtime.h"
#include "prime_search.cuh"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <iostream>
#include <mma.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>

// #define DEBUG_PERMUTE
#define DEBUG_PRINT 
// #define DEBUG_SORT
// #define DEBUG_RANDOM

#define SIZE 1024

// #define TOTAL_PERMUTATIONS 1000000 
#define TOTAL_PERMUTATIONS 10000000000
// #define TOTAL_PERMUTATIONS 10
#define CHECK_DONE_PERMUTATIONS 1000000

using namespace nvcuda;
using namespace std;

__global__ void prime_search(int* data, int size, int* output) {
    output[threadIdx.x] = data[threadIdx.x] + 1;

    return;
}

// __device__ long fast_size_lower_bound(int* data, long lower_bound) {

// }


dim3 KernelManagerPrimeSearch::calculateGrid(int n, int threadsPerBlock) {
    // return dim3((INNER_DIM + threadsPerBlock - 1) / threadsPerBlock);
    return dim3(n);
}

float KernelManagerPrimeSearch::launchKernel(int* data, int* output) {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    int smCount = 0;
    for (int i = 0; i < deviceCount; i = i + 1) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        smCount += prop.multiProcessorCount;
    }

    #ifdef DEBUG_PRINT
    printf("Number of SMs: %d\n", smCount);
    #endif

    int numBlocks = smCount * 2048 / SIZE;
    printf("Number of blocks: %d\n", numBlocks);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int threadsPerBlock = SIZE;
    dim3 grid = calculateGrid(numBlocks, threadsPerBlock);
    dim3 block(threadsPerBlock);
    #ifdef DEBUG_PRINT
    printf("Grid dimensions: %d x %d x %d\n", grid.x, grid.y, grid.z);
    #endif

    // Record start time
    hipEventRecord(start);

    // Launch kernel
    prime_search<<<1, block>>>(data, SIZE, output);
    // prime_search<<<grid, block>>>(data, SIZE, output);

    // Record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);


    return milliseconds;
}